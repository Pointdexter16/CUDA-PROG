#include<hip/hip_runtime.h>

#include<stdio.h>
#include<time.h>

#define N 16


void initialize(float *a,int n,int m){
    for(int y=0;y<n;y++){
        for(int x=0;x<m;x++){
            a[y*m+x]=2;
        }
    }
}

void initialize_zero(float *a,int n,int m){
    for(int y=0;y<n;y++){
        for(int x=0;x<m;x++){
            a[y*m+x]=0;
        }
    }
}

void display(float *a,int n,int m){
    for(int y=0;y<n;y++){
        for(int x=0;x<m;x++){
            printf("%f ",a[y*m+x]);
        }
        printf("\n");
    }
    printf("\n");
}

__global__
void MatrixMull(float* a, int N0, int M0, float* b, int N1, int M1, float* c){
    int i=blockIdx.y*blockDim.y + threadIdx.y;
    int j=blockIdx.x*blockDim.x + threadIdx.x;
    float result=0;
    if((i<N0)&&(j<M1)){
        for(int k=0;k<N;k++){
            result+=a[i*N + k]*b[k*N + j];
        }
        c[i*N + j]=result;        
    }
}


int main(){ 
    clock_t start,end;
    double Etime;
    float *a,*b,*c;
    size_t byte=(16*16)*(sizeof(float));
    a=(float*)malloc(byte);
    b=(float*)malloc(byte);
    c=(float*)malloc(byte);
    initialize(a,16,N);
    initialize(b,N,16);
    initialize_zero(c,16,16);
    float *d_a,*d_b,*d_c;
    hipMalloc(&d_a,byte);
    hipMalloc(&d_b,byte);
    hipMalloc(&d_c,byte);
    hipMemcpy( d_a, a, byte, hipMemcpyHostToDevice);
    hipMemcpy( d_b, b, byte, hipMemcpyHostToDevice);
    hipMemcpy( d_c, c, byte, hipMemcpyHostToDevice);
    dim3 blockDim(8,8,1);
    dim3 griddim(8,8,1);
    start=clock();
    MatrixMull<<< griddim, blockDim >>>(d_a, 16, N, d_b, 16, N, d_c);
    end=clock ();
    hipMemcpy(c,d_c, byte, hipMemcpyDeviceToHost);
    Etime=((double)(end-start))/CLOCKS_PER_SEC;
    printf("execution time: %lf",Etime);
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;   
}

#include<stdio.h>
#include<hip/hip_runtime.h>

#include<time.h>

#define N 256*9999

void ini(double *A,double val){
    for(int i=0;i<N;i++){
        A[i]=val;
    }
}

__global__ 
void addVector(double* a,double* b, double *c){
    int i=blockDim.x * blockIdx.x + threadIdx.x;
    if(i<N){
        c[i]=a[i]+b[i];
    }
}

int main(){

    clock_t start,end;
    double time;

    size_t bytes = N*sizeof(double);

    double *A = (double*)malloc(bytes);
    double *B = (double*)malloc(bytes);
    double *C = (double*)malloc(bytes);
    
    ini(A,1);
    ini(B,2);

    double *d_a,*d_b,*d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    hipMemcpy(d_a, A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, B, bytes, hipMemcpyHostToDevice);

    int thread=256;
    int block=ceil(float(N)/256);

    start=clock();
    addVector<<< block, thread >>>(d_a,d_b,d_c);
    end=clock();

    hipMemcpy(C, d_c, bytes, hipMemcpyDeviceToHost);

    printf("%lf + %lf=%lf",A[0],B[0],C[0]);

    free(A);
    free(B);
    free(C);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    printf("\n---------------------------\n");
	printf("__SUCCESS__\n");
	printf("---------------------------\n");
	printf("N                 = %d\n", N);
	printf("Threads Per Block = %d\n", thread);
	printf("Blocks In Grid    = %d\n", block);
	printf("---------------------------\n\n");

    time=(double(end-start))/CLOCKS_PER_SEC;
    printf("execution time: %lf",time);
    return 0;
}